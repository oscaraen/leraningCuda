﻿
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>


#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

const int threadsPerBlock = 4;
// kernel que hace la suma 
__global__ void sumarVectores(float* vector1, float* vector2, int tam) {
    int tid = threadIdx.x + blockIdx.x*blockDim.x; // pendiente saber como cambio blockDIm
    while (tid < tam) {
        printf("threadIdx: %d, blockIdx:%d, blockDim:%d, gridDim:%d tid: %d\n", threadIdx.x, blockIdx.x, blockDim.x, gridDim.x, tid);
        vector2[tid] += vector1[tid];
        tid += blockDim.x * gridDim.x; // el truco para hacer vectores muy largos o shared memory es sumarle blockDim+gridDim
    }
}


// Kernel para hacer producto punto con shared memory
__global__ void productoPunto(float* vector1, float* vector2, int tam, float * output) {
    __shared__ float cache[threadsPerBlock]; // creo una caché con thre
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    // parte 1: 
    while (tid < tam) {
        temp += vector1[tid] * vector2[tid];
        printf("tid: %d, temp is: %.2f\n", tid, temp); //primera ejecución, para vector arbitrario hay saltos de blockDim*gridDim
        tid += blockDim.x * gridDim.x;
    }

    // set the cache values
    cache[cacheIndex] = temp;
    // wait for all threads end
    __syncthreads();
    // para las reducciones el número de threads por bloque debe ser potencia de 2 para poder hacerla efectiva sin que sobren ni falten elementos
    int i = blockDim.x / 2; // blockDim dice cuantos threads per block hay
    while (i != 0) {
        float testVariable = 0;
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        testVariable = cache[cacheIndex];
        printf("cacheindex: %.2f\n", testVariable);
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        output[blockIdx.x] = cache[0]; // al final en cada 16th voy a tener la suma de todos los demás, hay que sumar esos


}

/** 
* Thread cooperation example:
* 
* Siguiendo el libro cuda_by_example a ver si es posible usar shared memory para alguna operación
*/
int main()
{
    // En cuda tenemos hiulos que hacen copias en paralelo de un código (kernel) (bloques)
    // Cuda permite que estos bloques se separen en threads
    // en el llamado se manda <<<bloques, hilos>>>

    float *vector1;
    float* vector2;

    vector1 = (float*)malloc(sizeof(float) * 500);
    vector2 = (float*)malloc(sizeof(float) * 500);
    for (int i = 0; i < 500; i++) {
        // Llenar los vectores con números cualquiera
        vector1[i] =1;
        vector2[i] = 1;
    }
    cout << vector1[0] << " v2: " << vector2[0] << endl;

    // reservar memoria en cuda
    float* d_vector1, * d_vector2, *d_vector3;
    hipMalloc(&d_vector1, sizeof(float) * 500);
    hipMalloc(&d_vector2, sizeof(float) * 500);
    hipMalloc(&d_vector3, sizeof(float) * 500);
    // copiar datos a cuda
    hipMemcpy(d_vector1, vector1, sizeof(float) * 500, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, vector2, sizeof(float) * 500, hipMemcpyHostToDevice);

    productoPunto <<<128, 128>>> (d_vector1, d_vector2, 100, d_vector3);

    hipMemcpy(vector1, d_vector3, sizeof(float) * 500, hipMemcpyDeviceToHost);

    for (int i = 0; i < 100; i++) {
        cout << vector1[i] << ", ";
    }
    cout << endl;


    cout << "Si lees esto, todo salio bien" << endl;

    
    return 0;
}

